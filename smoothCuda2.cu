#include "hip/hip_runtime.h"
/*
 SmoothSeq SmoothSeq.cpp `pkg-config --cflags --libs opencv`
	./SmoothSeq image_in type_img image_out
	type_img -> 0 = GRAYSCALE
	type_img -> 1 = COLOR
*/
	
#include "opencv2/core/core.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
#include <iostream>
#include <vector>
#include <cmath>
#include <sys/time.h>


using namespace cv;
using namespace std;

//Função que calcula a média de uma "matriz" 5x5 a partir de uma dada posição
__global__ void smooth( unsigned char *entrada,unsigned char *saida, int n_linhas, int n_colunas ) {
    //Calcula a posição no vetor (id_bloco * total_blocos + id_thread)
    int posicao = blockIdx.x * blockDim.x + threadIdx.x;
    //Se a posição não é maior que o limite da imagem original...
    if(posicao < (n_linhas)*(n_colunas)) {
        //soma o valor da região 5x5 em torno no pixel
        saida[posicao] =entrada[posicao]+
                        entrada[posicao+(n_colunas+4)]+
                        entrada[posicao+(2*(n_colunas+4))]+
                        entrada[posicao+(3*(n_colunas+4))]+
                        entrada[posicao+(4*(n_colunas+4))]+
                        entrada[posicao+1]+
                        entrada[posicao+(n_colunas+4)+1]+
                        entrada[posicao+(2*(n_colunas+4))+1]+
                        entrada[posicao+(3*(n_colunas+4))+1]+
                        entrada[posicao+(4*(n_colunas+4))+1]+
                        entrada[posicao+2]+
                        entrada[posicao+(n_colunas+4)+2]+
                        entrada[posicao+(2*(n_colunas+4))+2]+
                        entrada[posicao+(3*(n_colunas+4))+2]+
                        entrada[posicao+(4*(n_colunas+4))+2]+
                        entrada[posicao+3]+
                        entrada[posicao+(n_colunas+4)+3]+
                        entrada[posicao+(2*(n_colunas+4))+3]+
                        entrada[posicao+(3*(n_colunas+4))+3]+
                        entrada[posicao+(4*(n_colunas+4))+3]+
                        entrada[posicao+4]+
                        entrada[posicao+(n_colunas+4)+4]+
                        entrada[posicao+(2*(n_colunas+4))+4]+
                        entrada[posicao+(3*(n_colunas+4))+4]+
                        entrada[posicao+(4*(n_colunas+4))+4];
        //calcula a média
        saida[posicao] = saida[posicao]/25;
    }
}

int main(int argc, char *argv[]) {
	//diz se a imagem é grayscale or color
	int tipo_img = atoi(argv[2]);
	//arquivo de entrada
	const char *fileIn, *fileOut;
	
	//numero maximo de threads da placa do andromeda
    int nthreads = 1024;

    float nb;
    int numBlocks;


	//matriz com a imagem de entrada
	Mat in;
	//matriz que receberá a imagem de saida
	Mat *out;

	//le o nome da imagem
	fileIn = argv[1];
	fileOut = argv[3];
	//le e salva a imagem na matriz
	if(tipo_img == 0) {
		in = imread(fileIn, CV_LOAD_IMAGE_GRAYSCALE);
	} else if(tipo_img == 1) {
		in = imread(fileIn, CV_LOAD_IMAGE_COLOR);
	} else {
		cout << "Tipo de imagem nao suportado" << endl;
		return -1;
	}
	//caso nao consegui abrir a imagem
	if (in.empty()) {
		cout << "Nao foi possivel abrir a  imagem: " << endl;
		return -1;
	}

    //numero de blocos é o total de pixels dividido pelo total de threads
    nb = (in.size().width*in.size().height)/nthreads;
    cout << nb << endl;
    //O cast trunca o ponto flutuante, por isso soma-se 1
    numBlocks = (int) (nb + 1.0);
	cout << numBlocks << endl;
	unsigned char *original,*final;
	
    //Malloc especial do CUDA, para os vetores originais e de saída
    //Estes vetores são passados às funções que serão calculadas pela
    //placa de vídeo
    
    int *height, *width;
    hipMalloc(&original, (in.size().width + 4)*(in.size().height + 4));
    hipMalloc(&final, in.size().width*in.size().height);
    hipMalloc(&height, sizeof(int));
    hipMalloc(&width, sizeof(int));	
    int l_height = in.size().height, l_width = in.size().width;
	//pegar o tempo de inicio
	struct timeval inicio, fim;
    gettimeofday(&inicio,0);
    
    hipMemcpy(original, in.data,( in.size().width+4)*(in.size().height+4), hipMemcpyHostToDevice);
	hipMemcpy(height, &l_height, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(width, &l_width, sizeof(int), hipMemcpyHostToDevice);
	smooth<<<numBlocks,nthreads>>>(original,final, *height, *width);
	
	out = new Mat(in.size().height, in.size().width, CV_8U, 1);
	hipMemcpy(out->data, final, (out->size().width)*(out->size().height),hipMemcpyDeviceToHost);
	
	//pega o tempo de fim, faz a diferença e imprime na tela
	gettimeofday(&fim,0);
    float speedup = (fim.tv_sec + fim.tv_usec/1000000.0) - (inicio.tv_sec + inicio.tv_usec/1000000.0);
    cout << speedup << endl;
	imwrite(fileOut, *out);
	in.release();
	out->release();
    hipFree(original);
    hipFree(final);
    return 0;
}


//ssh grupo18b@halley.lasdpc.icmc.usp.br -p 22200
//grupo18b#raul


