#include "hip/hip_runtime.h"
/*SmoothSeq SmoothSeq.cpp `pkg-config --cflags --libs opencv`
	./SmoothSeq image_in type_img image_out
	type_img -> 0 = GRAYSCALE
	type_img -> 1 = COLOR
*/
	
#include "opencv2/core/core.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
#include <iostream>
#include <vector>
#include <cmath>
#include <sys/time.h>
#include <math.h>

using namespace cv;
using namespace std;

//Função que calcula a média de uma "matriz" 5x5 a partir de uma dada posição
__global__ void smooth( unsigned char *entrada,unsigned char *saida, int n_linhas, int n_colunas ) {
    //Calcula a posição no vetor (id_bloco * total_blocos + id_thread)
    int posicao = blockIdx.x * blockDim.x + threadIdx.x;
    //Se a posição não é maior que o limite da imagem original...
    if(posicao < (n_linhas)*(n_colunas)) {
        //soma o valor da região 5x5 em torno no pixel
        saida[posicao] =entrada[posicao]+
                        entrada[posicao+(n_colunas+4)]+
                        entrada[posicao+(2*(n_colunas+4))]+
                        entrada[posicao+(3*(n_colunas+4))]+
                        entrada[posicao+(4*(n_colunas+4))]+
                        entrada[posicao+1]+
                        entrada[posicao+(n_colunas+4)+1]+
                        entrada[posicao+(2*(n_colunas+4))+1]+
                        entrada[posicao+(3*(n_colunas+4))+1]+
                        entrada[posicao+(4*(n_colunas+4))+1]+
                        entrada[posicao+2]+
                        entrada[posicao+(n_colunas+4)+2]+
                        entrada[posicao+(2*(n_colunas+4))+2]+
                        entrada[posicao+(3*(n_colunas+4))+2]+
                        entrada[posicao+(4*(n_colunas+4))+2]+
                        entrada[posicao+3]+
                        entrada[posicao+(n_colunas+4)+3]+
                        entrada[posicao+(2*(n_colunas+4))+3]+
                        entrada[posicao+(3*(n_colunas+4))+3]+
                        entrada[posicao+(4*(n_colunas+4))+3]+
                        entrada[posicao+4]+
                        entrada[posicao+(n_colunas+4)+4]+
                        entrada[posicao+(2*(n_colunas+4))+4]+
                        entrada[posicao+(3*(n_colunas+4))+4]+
                        entrada[posicao+(4*(n_colunas+4))+4];
        //calcula a média
        saida[posicao] = saida[posicao]/25;
    }
}

int main(int argc, char *argv[]) {
	//diz se a imagem é grayscale or color
	int tipo_img = atoi(argv[2]);
	//arquivo de entrada
	const char *fileIn, *fileOut;
	
	//numero maximo de threads da placa do andromeda
    int nthreads = 1024;

    int numBlocks;


	//matriz com a imagem de entrada
	Mat in;
	//matriz que receberá a imagem de saida
	Mat *out;

	//le o nome da imagem
	fileIn = argv[1];
	fileOut = argv[3];
	//le e salva a imagem na matriz
	if(tipo_img == 0) {
		in = imread(fileIn, CV_LOAD_IMAGE_GRAYSCALE);
	} else if(tipo_img == 1) {
		in = imread(fileIn, CV_LOAD_IMAGE_COLOR);
	} else {
		cout << "Tipo de imagem nao suportado" << endl;
		return -1;
	}
	//caso nao consegui abrir a imagem
	if (in.empty()) {
		cout << "Nao foi possivel abrir a  imagem: " << endl;
		return -1;
	}
    int l_height = in.size().height, l_width = in.size().width;

    //numero de blocos é o total de pixels dividido pelo total de threads
    numBlocks = ceil((l_height*l_width)/nthreads);

	unsigned char *original,*saida;
  
    //Malloc especial do CUDA, para os vetores originais e de saída
    //Estes vetores são passados às funções que serão calculadas pela
    //placa de vídeo
    

    hipMalloc(&original, (l_width + 4) * (l_height + 4));
    hipMalloc(&saida, l_width * l_height);

	//pegar o tempo de inicio
    
    struct timeval inicio, fim;
    gettimeofday(&inicio,0);
    
    hipMemcpy(original, in.data,l_width * l_height, hipMemcpyHostToDevice);

	smooth<<<numBlocks,nthreads>>>(original, saida, l_height, l_width);
	
	out = new Mat(l_height, l_width, CV_8U, 1);
	hipMemcpy(out->data, saida, l_width*l_height,hipMemcpyDeviceToHost);
	
	//pega o tempo de fim, faz a diferença e imprime na tela
	gettimeofday(&fim,0);
    float speedup = (fim.tv_sec + fim.tv_usec/1000000.0) - (inicio.tv_sec + inicio.tv_usec/1000000.0);
    cout << speedup << endl;
	imwrite(fileOut, *out);
	in.release();
	out->release();
    hipFree(original);
    hipFree(saida);

    return 0;
}
